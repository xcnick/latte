#include <hip/hip_runtime.h>
#include "latte/util/math_functions.h"

#include "latte/common.h"
#include "latte/util/device_alternate.h"

namespace latte {

void latte_gpu_memcpy(const size_t N, const void *X, void *Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));
  }
}

void latte_gpu_memset(const size_t N, const int alpha, void *X) {
  CUDA_CHECK(hipMemset(X, alpha, N));  // NOLINT
}

template <>
void latte_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
                           const CBLAS_TRANSPOSE TransB, const int M,
                           const int N,  int K, const float alpha,
                           const float *A, const float *B, const float beta,
                           float *C) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Latte::cublas_handle(), cuTransB, cuTransA, N, M, K,
                           &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void latte_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
                            const CBLAS_TRANSPOSE TransB, const int M,
                            const int N, const int K, const double alpha,
                            const double *A, const double *B, const double beta,
                            double *C) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Latte::cublas_handle(), cuTransB, cuTransA, N, M, K,
                           &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void latte_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
                           const int N, const float alpha, const float *A,
                           const float *x, const float beta, float *y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Latte::cublas_handle(), cuTransA, N, M, &alpha, A, N,
                           x, 1, &beta, y, 1));
}

template <>
void latte_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
                            const int N, const double alpha, const double *A,
                            const double *x, const double beta, double *y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Latte::cublas_handle(), cuTransA, N, M, &alpha, A, N,
                           x, 1, &beta, y, 1));
}

template <>
void latte_gpu_axpy<float>(const int N, const float alpha, const float *X,
                           float *Y) {
  CUBLAS_CHECK(hipblasSaxpy(Latte::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void latte_gpu_axpy<double>(const int N, const double alpha, const double *X,
                            double *Y) {
  CUBLAS_CHECK(hipblasDaxpy(Latte::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void latte_gpu_dot<float>(const int n, const float *x, const float *y,
                          float *out) {
  CUBLAS_CHECK(hipblasSdot(Latte::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void latte_gpu_dot<double>(const int n, const double *x, const double *y,
                           double *out) {
  CUBLAS_CHECK(hipblasDdot(Latte::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void latte_gpu_asum<float>(const int n, const float *x, float *y) {
  CUBLAS_CHECK(hipblasSasum(Latte::cublas_handle(), n, x, 1, y));
}

template <>
void latte_gpu_asum<double>(const int n, const double *x, double *y) {
  CUBLAS_CHECK(hipblasDasum(Latte::cublas_handle(), n, x, 1, y));
}

template <>
void latte_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Latte::cublas_handle(), N, &alpha, X, 1));
}

template <>
void latte_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Latte::cublas_handle(), N, &alpha, X, 1));
}

}  // namespace latte